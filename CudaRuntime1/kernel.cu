#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void myFirstKernel() {
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    int globalThreadId = blockId * blockDim.x + threadId;
    printf("Block %d, Thread %d, Global Thread %d\n", blockId, threadId, globalThreadId);
}

int main() {
    myFirstKernel << <4, 8 >> > ();
    hipDeviceSynchronize();
    return 0;
}

