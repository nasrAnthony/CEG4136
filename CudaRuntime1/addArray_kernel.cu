#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>



__global__ void kernel_add_arrays(const int* a, const int* b, int* c) {
	int index = threadIdx.x;
	c[index] = a[index] + b[index];
}

int main() {
	const int arraySize = 5;
	int a[arraySize] = { 1, 2, 3, 4, 5 };
	int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };


	int* dev_a, * dev_b, * dev_c; // init pointer to the device for each array

	//allocate memory space on device
	hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	hipMalloc((void**)&dev_b, arraySize * sizeof(int));
	hipMalloc((void**)&dev_c, arraySize * sizeof(int));

	//copy contents of arrays into allocated memory on device
	hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, arraySize * sizeof(int), hipMemcpyHostToDevice);


	//call kernel
	kernel_add_arrays << < 1, 5 >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < arraySize; i++) {
		printf("%d\n", c[i]);
	}

	return 0;
}

