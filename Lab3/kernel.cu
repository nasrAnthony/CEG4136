#include "hip/hip_runtime.h"
﻿#include <iostream> 
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

//#define T 2 //32 //Tile size 32
#define T 32

void buildMatrix(int* start, int& size) { //helper function to init the matrices before multiplication. 
	for (int i = 0; i < size; i++) {
		*(start + i) = rand() % 10;
	};
}

void printMatrix(int* start, int& size) { //helper function to showcase matrix 
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			std::cout << *(start + i * size + j) << " "; //convert 2D index to 1D for printing. 
		}
		std::cout << std::endl;
	}
}

//Kernel implementation
__global__ void matrixMultKernel(int* A, int* B, int* C, int size) {
	//shared memory setup
	__shared__ int Ashm[T][T];
	__shared__ int Bshm[T][T];
	//calculate index
	int row = blockIdx.y * T + threadIdx.y;
	int col = blockIdx.x * T + threadIdx.x;

	int val = 0;

	for (int i = 0; i < size / T; i++) {
		Ashm[threadIdx.y][threadIdx.x] = A[row * size + i * T + threadIdx.x];
		Bshm[threadIdx.y][threadIdx.x] = B[(i * T + threadIdx.y) * size + col];
		__syncthreads();
		for (int j = 0; j < T; j++) {
			val += Ashm[threadIdx.y][j] * Bshm[j][threadIdx.x];
		}
		__syncthreads();
	}
	C[row * size + col] = val;
}

int main() {

	//int N = 8;//= 1024;
	int N = 1024;
	int totalSize = N * N;
	//fetch total size of array in bytes
	int size = N * N * sizeof(int);
	//allocate memory on host. 
	int* hostA = (int*)malloc(size);//cast pointer returned from malloc to type int* || operand A
	int* hostB = (int*)malloc(size);//cast pointer returned from malloc to type int* || operand B
	int* hostC = (int*)malloc(size);//cast pointer returned from malloc to type int* || result  C

	//build the matrices with helper function
	buildMatrix(hostA, totalSize); //init matrix A
	buildMatrix(hostB, totalSize); //init matrix B

	//Print matrices A and B
	//std::cout << "Matrix A:" << std::endl;
	//printMatrix(hostA, N);
	//std::cout << "\nMatrix B:" << std::endl;
	//printMatrix(hostB, N);

	//allocate device memory
	int* devA; int* devB; int* devC;
	hipMalloc((void**)&devA, size);
	hipMalloc((void**)&devB, size);
	hipMalloc((void**)&devC, size);

	//copy the matrices to device
	hipMemcpy(devA, hostA, size, hipMemcpyHostToDevice);
	hipMemcpy(devB, hostB, size, hipMemcpyHostToDevice);

	//setup block size
	dim3 blockDim(T, T); //32x32
	dim3 gridDim((N + T - 1) / T, (N + T - 1) / T);


	matrixMultKernel << <gridDim, blockDim >> > (devA, devB, devC, N); //call kernel
	hipDeviceSynchronize();

	//copy result back to device
	hipMemcpy(hostC, devC, size, hipMemcpyDeviceToHost);

	//Print matrix C
	//std::cout << "\nResult Matrix C (A * B):" << std::endl;
	//printMatrix(hostC, N);

	//free up space on device
	hipFree(devA); hipFree(devB); hipFree(devC);
	//free up space on host
	free(hostA), free(hostB); free(hostC);
	return 0;

}
