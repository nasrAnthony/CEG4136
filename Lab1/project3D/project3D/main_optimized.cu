#include "hip/hip_runtime.h"
#include <GL/glut.h>
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <random>    // Pour std::shuffle et std::mt19937 // For std::shuffle and std::mt19937
#include <algorithm> // Pour std::shuffle // For std::shuffle
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N 1000  // Taille de la grille // Grid size
#define BURN_DURATION 5000  // Dur�e de combustion d'un arbre en millisecondes (5 secondes) // Tree burning duration in milliseconds (5 seconds)
#define FIRE_START_COUNT 100  // Nombre initial d'incendies // Initial number of fire locations

// Utilisation de vecteurs pour g�rer la m�moire // Using vectors to manage memory
std::vector<std::vector<int>> forest(N, std::vector<int>(N, 0));
std::vector<std::vector<int>> burnTime(N, std::vector<int>(N, 0));
bool allBurnedOut = true;  // Indicateur pour v�rifier si tous les feux sont �teints // Flag to check if all fires are out

int simulationDuration = 60000;  // Dur�e de la simulation (60 secondes) // Simulation duration (60 seconds)
int startTime = 0;  // Temps de d�part en millisecondes // Start time in milliseconds
int elapsedTime = 0;  // Temps �coul� // Elapsed time
float spreadProbability = 0.3f;  // Probabilit� que le feu se propage � un arbre voisin // Probability that fire spreads to a neighboring tree

bool isPaused = false;  // Indicateur de pause // Pause indicator
int pauseStartTime = 0;  // Temps de d�but de la pause // Start time of pause

float zoomLevel = 1.0f;  // Niveau de zoom // Zoom level
float offsetX = 0.0f, offsetY = 0.0f;  // D�calage horizontal et vertical pour le d�placement // Horizontal and vertical offset for movement
float moveSpeed = 0.05f;  // Vitesse de d�placement de la vue // View movement speed

bool dragging = false;  // Indicateur de glisser-d�poser avec la souris // Mouse drag indicator
int lastMouseX, lastMouseY;  // Derni�re position de la souris lors du clic // Last mouse position when clicked

// Fonction pour initialiser la for�t // Function to initialize the forest
void initializeForest() {
    // Initialisation de la for�t avec 50% d'arbres // Initializing the forest with 50% trees
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            forest[i][j] = rand() % 2;  // 50% d'arbres (1), 50% vide (0) // 50% trees (1), 50% empty space (0)
            burnTime[i][j] = 0;  // Aucun arbre ne br�le au d�part // No tree is burning at the start
        }
    }

    // Liste de positions disponibles pour allumer les feux // List of available positions to start fires
    std::vector<std::pair<int, int>> availablePositions;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (forest[i][j] == 1) {  // Ajouter les positions avec des arbres dans la liste // Add positions with trees to the list
                availablePositions.push_back({ i, j });
            }
        }
    }

    // M�langer les positions disponibles pour une distribution plus uniforme // Shuffle the available positions for a more uniform distribution
    std::random_device rd;  // G�n�rateur de nombres al�atoires bas� sur l'impl�mentation du syst�me // Random number generator based on system implementation
    std::mt19937 g(rd());   // G�n�rateur de nombres pseudo-al�atoires bas� sur Mersenne Twister // Mersenne Twister-based pseudo-random number generator
    std::shuffle(availablePositions.begin(), availablePositions.end(), g);

    // Allumer des feux de mani�re uniforme sur la grille // Ignite fires uniformly across the grid
    for (int fire = 0; fire < FIRE_START_COUNT && !availablePositions.empty(); fire++) {
        int fireX = availablePositions[fire].first;
        int fireY = availablePositions[fire].second;

        forest[fireX][fireY] = 2;  // Allumer l'arbre en feu // Ignite the tree
        burnTime[fireX][fireY] = BURN_DURATION;  // D�finir le temps de combustion // Set the burn duration
    }

    startTime = glutGet(GLUT_ELAPSED_TIME);  // R�initialiser le temps de d�part // Reset start time
    elapsedTime = 0;  // R�initialiser le temps �coul� // Reset elapsed time
    isPaused = false;  // Fin de la pause // End of pause
}

// Fonction d'initialisation OpenGL // OpenGL initialization function
void initGL() {
    glClearColor(1.0, 1.0, 1.0, 1.0);  // Couleur de fond blanche // White background color
    glEnable(GL_DEPTH_TEST);  // Activer le test de profondeur // Enable depth test
}

// Fonction pour dessiner la grille // Function to draw the grid
void drawForest() {
    float cellSize = 2.0f / N;  // Taille de chaque cellule ajust�e par la taille N // Adjusted cell size based on grid size N

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            // Choisir la couleur en fonction de l'�tat de la cellule // Set color based on the state of the cell
            if (forest[i][j] == 0 && burnTime[i][j] == 0) {
                glColor3f(0.8f, 0.8f, 0.8f);  // Espace vide (gris) // Empty space (gray)
            }
            else if (forest[i][j] == 1) {
                glColor3f(0.0f, 1.0f, 0.0f);  // Arbre (vert) // Tree (green)
            }
            else if (forest[i][j] == 2) {
                glColor3f(1.0f, 0.0f, 0.0f);  // Arbre en feu (rouge) // Tree on fire (red)
            }
            else if (forest[i][j] == 3) {
                glColor3f(0.0f, 0.0f, 0.0f);  // Arbre br�l� (noir) // Burned tree (black)
            }

            // Dessiner la cellule // Draw the cell
            float x = -1.0f + j * cellSize;
            float y = -1.0f + i * cellSize;
            glBegin(GL_QUADS);
            glVertex2f(x, y);
            glVertex2f(x + cellSize, y);
            glVertex2f(x + cellSize, y + cellSize);
            glVertex2f(x, y + cellSize);
            glEnd();
        }
    }
}

__global__ void updateForestKernel(int* forest, int* burnTime, float* spreadProbability, bool* allBurnedFlag, int gridSize) {
    int gridDims = gridSize;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row = idx / gridDims;
    int col = idx % gridDims;

    // Initialize cuRAND state for this thread
    hiprandState state;
    hiprand_init((unsigned long long)clock() + idx, 0, 0, &state);

    // Check if the tree at this index is on fire
    if (forest[idx] == 2) {  // If the tree is on fire
        burnTime[idx] -= 200;  // Reduce burning time

        if (burnTime[idx] <= 0) {
            forest[idx] = 3;  // Tree has burned out
        }
        else {
            // Spread fire to top neighbor
            if (row > 0 && forest[(row - 1) * gridDims + col] == 1) {
                int randVal = hiprand(&state);
                if ((randVal / float(RAND_MAX)) < *spreadProbability) {
                    forest[(row - 1) * gridDims + col] = 2;
                    burnTime[(row - 1) * gridDims + col] = BURN_DURATION;
                }
            }

            // Spread fire to bottom neighbor
            if (row < gridDims - 1 && forest[(row + 1) * gridDims + col] == 1) {
                int randVal = hiprand(&state);
                if ((randVal / float(RAND_MAX)) < *spreadProbability) {
                    forest[(row + 1) * gridDims + col] = 2;
                    burnTime[(row + 1) * gridDims + col] = BURN_DURATION;
                }
            }

            // Spread fire to left neighbor
            if (col > 0 && forest[row * gridDims + (col - 1)] == 1) {
                int randVal = hiprand(&state);
                if ((randVal / float(RAND_MAX)) < *spreadProbability) {
                    forest[row * gridDims + (col - 1)] = 2;
                    burnTime[row * gridDims + (col - 1)] = BURN_DURATION;
                }
            }

            // Spread fire to right neighbor
            if (col < gridDims - 1 && forest[row * gridDims + (col + 1)] == 1) {
                int randVal = hiprand(&state);
                if ((randVal / float(RAND_MAX)) < *spreadProbability) {
                    forest[row * gridDims + (col + 1)] = 2;
                    burnTime[row * gridDims + (col + 1)] = BURN_DURATION;
                }
            }
        }
        // If any tree is still burning, the flag should be set to false
        if (forest[idx] == 2) {
            *allBurnedFlag = false;
        }
    }
}



// Fonction pour mettre � jour la for�t et la propagation du feu // Function to update the forest and fire propagation
void updateForest() {
    if (isPaused) {  // Si la simulation est en pause, r�initialiser la for�t apr�s la pause // If the simulation is paused, reset the forest after the pause
        if (glutGet(GLUT_ELAPSED_TIME) - pauseStartTime >= 3000) {
            initializeForest();  // R�initialiser la for�t apr�s 3 secondes // Reset the forest after 3 seconds
        }
        return;
    }


    std::vector<std::vector<int>> newForest = forest;  // Copie la for�t actuelle // Copy the current forest

    //create pointer initial pointer to vector.
    int* dev_forest;
    int* dev_burn_time;
    bool* dev_allBurnedOut;
    float* dev_SpreadProb;

    std::vector<int> flatForest(N * N);
    std::vector<int> flatBurnTime(N * N);
    for (int i = 0; i < N; i++) {
        std::copy(forest[i].begin(), forest[i].end(), flatForest.begin() + i * N);
        std::copy(burnTime[i].begin(), burnTime[i].end(), flatBurnTime.begin() + i * N);
    }
    //allocate memory space on device to hold forest.
    hipMalloc((void**)&dev_forest, N * N * sizeof(int));
    //allocate memory space on device to hold burn time grid.
    hipMalloc((void**)&dev_burn_time, N * N * sizeof(int));
    //allocate memory space on device to hold the bool
    hipMalloc((void**)&dev_allBurnedOut, sizeof(int));
    //smth  
    hipMalloc((void**)&dev_SpreadProb, sizeof(float));


    //copy the new flat grids to the device memory
    hipMemcpy(dev_forest, flatForest.data(), N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_burn_time, flatBurnTime.data(), N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_allBurnedOut, &allBurnedOut, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(dev_SpreadProb, &spreadProbability, sizeof(float), hipMemcpyHostToDevice);

    //define parameters for further optimization/testing. 
    int totalThreads = N * N;
    int threadsPerBlock = 512;  // This is a common maximum for many GPUs
    int numBlocks = (totalThreads + threadsPerBlock - 1) / threadsPerBlock;

    bool allBurnedOut = true;

    //call kernel from host code
    updateForestKernel << < numBlocks, threadsPerBlock >> > (dev_forest, dev_burn_time, dev_SpreadProb, dev_allBurnedOut, N);

    hipDeviceSynchronize();
    hipMemcpy(flatForest.data(), dev_forest, N * N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(flatBurnTime.data(), dev_burn_time, N * N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&allBurnedOut, dev_allBurnedOut, sizeof(bool), hipMemcpyDeviceToHost);


    hipFree(dev_forest);
    hipFree(dev_burn_time);
    hipFree(dev_allBurnedOut);

    for (int i = 0; i < N; i++) {
        std::copy(flatForest.begin() + i * N, flatForest.begin() + (i + 1) * N, forest[i].begin());
        std::copy(flatBurnTime.begin() + i * N, flatBurnTime.begin() + (i + 1) * N, burnTime[i].begin());
    }

    if (allBurnedOut) {  // Si tous les feux sont �teints, mettre la simulation en pause // If all fires are out, pause the simulation
        isPaused = true;
        pauseStartTime = glutGet(GLUT_ELAPSED_TIME);
    }
}

// Fonction d'affichage // Display function
void display() {
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);  // Effacer le tampon de couleur et de profondeur // Clear color and depth buffer
    glLoadIdentity();  // R�initialiser la matrice mod�le-vue // Reset the model-view matrix
    glTranslatef(offsetX, offsetY, 0.0f);  // Appliquer le d�calage // Apply translation offset
    glScalef(zoomLevel, zoomLevel, 1.0f);  // Appliquer le zoom // Apply zoom
    drawForest();  // Dessiner la for�t // Draw the forest
    glutSwapBuffers();  // �changer les tampons pour afficher l'image // Swap buffers to display the image
}

// Fonction pour animer la simulation // Function to animate the simulation
void update(int value) {
    updateForest();  // Mettre � jour la for�t � chaque cycle // Update the forest at each cycle
    glutPostRedisplay();  // Demander un nouveau rendu // Request a new rendering
    glutTimerFunc(200, update, 0);  // Programmer la prochaine mise � jour dans 200 ms // Schedule the next update in 200 ms
}

// Gestion du clavier pour zoomer/d�zoomer et r�initialiser // Keyboard handling for zooming and resetting
void keyboard(unsigned char key, int x, int y) {
    switch (key) {
    case '+':
        zoomLevel *= 1.1f;  // Augmenter le niveau de zoom // Increase zoom level
        break;
    case '-':
        zoomLevel /= 1.1f;  // Diminuer le niveau de zoom // Decrease zoom level
        if (zoomLevel < 0.1f) zoomLevel = 0.1f;
        break;
    case 'r':  // Touche pour r�initialiser // Reset key
        zoomLevel = 1.0f;  // R�initialiser le zoom et le d�calage // Reset zoom and offset
        offsetX = 0.0f;
        offsetY = 0.0f;
        break;
    case 27:  // Touche �chap pour quitter // Escape key to quit
        exit(0);
    }
    glutPostRedisplay();  // Redessiner la sc�ne // Redraw the scene
}

// Gestion des touches fl�ch�es pour d�placer la vue // Arrow keys handling for moving the view
void specialKeys(int key, int x, int y) {
    switch (key) {
    case GLUT_KEY_UP:
        offsetY += moveSpeed / zoomLevel;  // D�placer la vue vers le haut // Move the view up
        break;
    case GLUT_KEY_DOWN:
        offsetY -= moveSpeed / zoomLevel;  // D�placer la vue vers le bas // Move the view down
        break;
    case GLUT_KEY_LEFT:
        offsetX += moveSpeed / zoomLevel;  // D�placer la vue vers la gauche // Move the view left
        break;
    case GLUT_KEY_RIGHT:
        offsetX -= moveSpeed / zoomLevel;  // D�placer la vue vers la droite // Move the view right
        break;
    }
    glutPostRedisplay();  // Redessiner la sc�ne // Redraw the scene
}

// Gestion de la souris pour d�placer la vue // Mouse handling for moving the view
void mouseMotion(int x, int y) {
    if (dragging) {
        offsetX += (x - lastMouseX) * moveSpeed / zoomLevel;  // Mettre � jour le d�calage horizontal // Update horizontal offset
        offsetY -= (y - lastMouseY) * moveSpeed / zoomLevel;  // Mettre � jour le d�calage vertical // Update vertical offset
        lastMouseX = x;
        lastMouseY = y;
        glutPostRedisplay();  // Redessiner la sc�ne // Redraw the scene
    }
}

// Fonction pour g�rer le clic de souris // Function to handle mouse clicks
void mouse(int button, int state, int x, int y) {
    if (button == GLUT_LEFT_BUTTON) {  // Si le bouton gauche de la souris est enfonc� // If the left mouse button is pressed
        if (state == GLUT_DOWN) {
            dragging = true;
            lastMouseX = x;
            lastMouseY = y;
        }
        else {
            dragging = false;
        }
    }
}

// Fonction principale // Main function
int main(int argc, char** argv) {
    srand(static_cast<unsigned>(time(NULL)));  // Initialiser le g�n�rateur de nombres al�atoires // Initialize random number generator
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(800, 800);
    glutCreateWindow("Simulation de feux de for�t/Forest Fire Simulation");  // Cr�er la fen�tre OpenGL // Create the OpenGL window

    initGL();
    initializeForest();

    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutSpecialFunc(specialKeys);
    glutMouseFunc(mouse);
    glutMotionFunc(mouseMotion);
    glutTimerFunc(200, update, 0);

    glutMainLoop();
    return 0;
}