#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream> 
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

using namespace std;

#define BDIMX 32 
#define BDIMY 32 

void buildMatrix(float* start, int& sizeX, int& sizeY) { //helper function to init the matrices before multiplication. 
    for (int i = 0; i < sizeY; i++) {
        for (int j = 0; j < sizeX; j++) {
            start[i * sizeX + j] = static_cast<float>(rand() % 10);
        }
    }
}

void printMatrix(float* matrix, int& sizeX, int& sizeY) { //helper function to showcase matrix 
    for (int i = 0; i < sizeY; i++) {
        for (int j = 0; j < sizeX; j++) {
            cout << matrix[i * sizeX + j] << " "; //convert 2D index to 1D for printing. 
        }
        cout << endl;
    }
}

__global__ void matrixTranspose(float* out, float* in, int nx, int ny) {

    // static shared memory. 
    __shared__ float tile[BDIMY][BDIMX + 1]; // Added padding

    // coordinate in original matrix
    unsigned int ix, iy, ti, to;
    ix = blockIdx.x * blockDim.x + threadIdx.x;
    iy = blockIdx.y * blockDim.y + threadIdx.y;

    #pragma unroll 
    for (int i = 0; i < BDIMY; i += 4) {
        int iyUnroll = iy + i;

        if (ix < nx && (threadIdx.y + i) < BDIMY && iyUnroll < ny) {
            // load data from global memory to shared memory. Writing in row-major order.
            tile[threadIdx.y + i][threadIdx.x] = in[(iyUnroll * nx) + ix];
        }
    }

    // thread synchronization
    __syncthreads();

    // thread index 
    unsigned int bidx, irow, icol;

    bidx = threadIdx.y * blockDim.x + threadIdx.x;

    irow = bidx / blockDim.y;
    icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    ix = blockIdx.y * blockDim.y + icol;
    iy = blockIdx.x * blockDim.x + irow;

    // linear global memory index for transposed matrix
    to = iy * ny + ix;

    // transpose with boundary test

    #pragma unroll
    for (int i = 0; i < BDIMY; i += 4) {
        int ixUnroll = ix + i;

        if (iy < nx && (irow + i) < BDIMY && ixUnroll < ny) {
            // store data to global memory from shared memory. Reading in column-major order.
            out[to + (i * ny)] = tile[icol][irow + i];
        }
    }

}

int main()
{
    int x = 1024;
    int y = 1024;


    int matrixSize = x * y;

    int matrixMemSize = matrixSize * sizeof(float);

    // Allocate memory on host
    float* host_matrix = (float*)malloc(matrixMemSize);
    float* host_matrix_transposed = (float*)malloc(matrixMemSize);

    // Build matrix
    buildMatrix(host_matrix, x, y);

    // Print matrix before transpose
    //cout << "Matrix before transpose:" << endl;
    // printMatrix(host_matrix, x, y);

    // Allocate device memory
    float* dev_matrix;
    float* dev_matrix_transposed;

    hipMalloc((void**)&dev_matrix, matrixMemSize);
    hipMalloc((void**)&dev_matrix_transposed, matrixMemSize);

    // Copy matrices to device
    hipMemcpy(dev_matrix, host_matrix, matrixMemSize, hipMemcpyHostToDevice);

    // Setup block size
    dim3 block(BDIMX, BDIMY);

    dim3 grid((x + BDIMX - 1) / BDIMX, (y + BDIMY - 1) / BDIMY);

    matrixTranspose << <grid, block >> > (dev_matrix_transposed, dev_matrix, x, y);
    hipDeviceSynchronize();

    // Copy result back to device
    hipMemcpy(host_matrix_transposed, dev_matrix_transposed, matrixMemSize, hipMemcpyDeviceToHost);

    cout << endl;

    // Print matrix after transpose
    //cout << "Matrix after transpose:" << endl;
    //printMatrix(host_matrix_transposed, y, x);

    // Free up space on device
    hipFree(dev_matrix);
    hipFree(dev_matrix_transposed);

    // Free up space on host
    free(host_matrix);
    free(host_matrix_transposed);

    return 0;
}
