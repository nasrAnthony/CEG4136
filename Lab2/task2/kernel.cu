#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdlib.h>

// Reduce warp divergence by using interleaved pairs instead of neighboring pairs. 
// Make interleaved pairs a template kernel.
template <typename T>
__global__ void interleavedPairsSum(T* i_arr_global, T* o_arr_global, int size)
{
    int tid = threadIdx.x;

	// Note the "* 2" for loop unrolling
    int idx = blockDim.x * blockIdx.x * 2 + tid; 

	T* i_data = blockIdx.x * blockDim.x * 2 + i_arr_global;

	// unrolling 2 data blocks
	if(idx + blockDim.x < size) {
		i_arr_global[idx] += i_arr_global[idx + blockDim.x];
	}
	
	// in-place reduction in global memory
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if (tid < stride) {
			i_data[tid] += i_data[tid + stride];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) {
		o_arr_global[blockIdx.x] = i_data[0];
	}
}

// Generate a generic array.
template <typename T>
void generateArray(T* arr, int size) {
    srand(time(NULL));

    for (int i = 0; i < size; i++) {

		if constexpr (std::is_integral<T>::value) {
			arr[i] = rand() % 11; // 0 to 10 inclusive.
		} 
		
		else if constexpr (std::is_floating_point<T>::value) {
			arr[i] = static_cast<T>(rand()) / static_cast<T>(RAND_MAX) * static_cast<T>(10); // 0.0 to 10.0 inclusive
 		}

    }

}

template <typename T>
void checkArray(T* arr, int size) {
	T sum = 0;
	for (int i = 0; i < size; i++) {
		sum += arr[i];
	}

	printf("Sum from Host: %d\n", sum);
}

int main()
{
    const int size = 512; // Max number for compatiblity.

	int grid = 1;
	int block = size;

	size_t bytes = size * sizeof(int);
	int *hostInputArray = (int *) malloc(bytes);
	int *hostOutputArray = (int *) malloc(sizeof(int));

    generateArray(hostInputArray, size);

	// Allocate Device Memory
	int *deviceInputArray = NULL;
	int *deviceOutputArray = NULL;
	hipMalloc((void **) &deviceInputArray, bytes);
	hipMalloc((void **) &deviceOutputArray, sizeof(int));

	hipMemcpy(deviceInputArray, hostInputArray, bytes, hipMemcpyHostToDevice);

	hipDeviceSynchronize();

    interleavedPairsSum<<<grid, block>>>(deviceInputArray, deviceOutputArray, size);
    
    hipDeviceSynchronize();

    // Copy the result back to host
	hipMemcpy(hostOutputArray, deviceOutputArray, sizeof(int), hipMemcpyDeviceToHost);

	int sum = hostOutputArray[0];

    printf("\nSum from Device: %d\n", sum);
	checkArray(hostInputArray, size);

    // Free device memory
    /// free host memory
	free(hostInputArray);
	free(hostOutputArray);
	// free device memory
	hipFree(deviceInputArray);
	hipFree(deviceOutputArray);

    return 0;
}

