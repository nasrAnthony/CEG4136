#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdlib.h>

__global__ void neighboredPairsNested(int *i_arr_global, int *o_arr_global, unsigned int size) {
    // set thread ID
    unsigned int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = i_arr_global + blockIdx.x*blockDim.x;
    int *odata = &o_arr_global[blockIdx.x];

    // stop condition
    if (size == 2 && tid == 0) {
        o_arr_global[blockIdx.x] = idata[0]+idata[1];
        return;
    }

    // nested invocation
    int offset = size >> 1;

    if(offset > 1 && tid < offset) {
        // in place reduction
        idata[tid] += idata[tid + offset];
    }

    // sync at block level
    __syncthreads();

    // nested invocation to generate child grids
    if(tid==0) {
        neighboredPairsNested<<<1, offset>>>(idata,odata,offset);

        // sync all child grids launched in this block
        hipDeviceSynchronize();
    }

    // sync at block level again
    __syncthreads();
}

void generateArray(int* arr, int size) {
    srand(time(NULL));

    for (int i = 0; i < size; i++) {
        arr[i] = rand() % 11; // 0 to 10 inclusive.
    }

}

void checkArray(int* arr, int size) {
	int sum = 0;
	for (int i = 0; i < size; i++) {
		sum += arr[i];
	}

	printf("Sum from Host: %d\n", sum);
}

int main()
{
    // Trial and error configuration based on Nvidia MX250 (Pascal Architecture).
    int threads = 256;
    int blocks = 256;
    int size = threads * blocks;

	dim3 block (threads, 1);
    dim3 grid  (blocks, 1);

	// allocate host memory
	int* hostInputArray = (int *) malloc(size * sizeof(int));
	int* hostOutputArray = (int *) malloc(grid.x * sizeof(int));

	generateArray(hostInputArray, size);

	// Allocate device memory
	int* deviceInputArray = NULL;
	int* deviceOutputArray = NULL;

	hipMalloc((void **) &deviceInputArray, size * sizeof(int));
	hipMalloc((void **) &deviceOutputArray, grid.x * sizeof(int));

	// Copy host memory to device memory
	hipMemcpy(deviceInputArray, hostInputArray, size * sizeof(int), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	// Launch Kernel
	neighboredPairsNested<<<grid, block>>>(deviceInputArray, deviceOutputArray, block.x);

	hipDeviceSynchronize();

	// Copy Device memory to Host memory
	hipMemcpy(hostOutputArray, deviceOutputArray, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();

	// Get and print sum of array elements.
	int sum = 0;

    for (int i = 0; i < grid.x; i++){ 
        sum += hostOutputArray[i];
    } 

	printf("Sum from Device: %d\n", sum);

	checkArray(hostInputArray, size);

	// Free host memory
	free(hostInputArray);
	free(hostOutputArray);

	// Free device memory
	hipFree(deviceInputArray);
	hipFree(deviceOutputArray);

    return 0;
}